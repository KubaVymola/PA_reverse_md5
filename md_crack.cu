#include "hip/hip_runtime.h"
#include <stdio.h>
#include "md5_mine.cu"

// __device__ const char alphabet[] = {
//     'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm',
//     'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z',
//     'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M',
//     'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z',
//     '0', '1', '2', '3', '4', '5', '6', '7', '8', '9'
// };

const char alphabet[] = {
    'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm',
    'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z',
    'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M',
    'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z',
    '0', '1', '2', '3', '4', '5', '6', '7', '8', '9'
};


__device__ const int alphabetSize = sizeof(alphabet) / sizeof(char);

/**
 * To copy output from the device to the host
 */
__device__ char * my_strcpy(char *dest, uint8_t *src, int length){
    for (int i = 0; i < length; i++) {
        dest[i] = (char)src[i];
    }

    dest[length] = 0;
    
    return dest;
}

__device__ int hex2int(char ch) {
    if (ch >= '0' && ch <= '9')
        return ch - '0';
    if (ch >= 'A' && ch <= 'F')
        return ch - 'A' + 10;
    if (ch >= 'a' && ch <= 'f')
        return ch - 'a' + 10;
    return -1;
}

__device__ bool compareTargetHash(const char * targetHash, uint8_t * digest) {
    for (int i = 0; i < 16; i++) {
        int hashIntVal = hex2int(targetHash[i * 2]) * 16 + hex2int(targetHash[i * 2 + 1]);

        if (hashIntVal != digest[i]) return false;
    }
    return true;
}

__device__ bool nextWord(int * indexes, int wordLength) {
    for (int i = wordLength - 1; i >= 6; i--) {
        indexes[i] = indexes[i] + 1;
        
        if (indexes[i] < alphabetSize) return true;

        indexes[i] = 0;
    }

    return false;
}

__device__ void indexesToChars(int * indexes, uint8_t * chars, int wordLength, char * alphabet) {
    for (int i = 0; i < wordLength; i++) {
        chars[i] = (uint8_t)alphabet[indexes[i]];
    }
}

__device__ void printArray(int * array, int length) {
    for (int i = 0; i < length; i++) {
        printf("%d,", array[i]);
    }

    printf("\n");
}

__device__ void printDigest(uint8_t * digest) {
    printf("Digest: ");
    
    for (int i = 0; i < 16; i++) {
        printf("%x,", digest[i]);
    }
    printf("\n");
}

__global__ void kernelCrackMd5(const char * targetHash, int wordLength, char * output, char * alphabet) {
    uint8_t digest[16];

    int * currentWord = (int *)malloc(sizeof(int) * wordLength);
    uint8_t * currentChars = (uint8_t *)malloc(sizeof(uint8_t) * wordLength + 1);
    currentChars[wordLength] = 0;

    // Get first 6 digits from coordinates in the grid
    int digit0 = blockIdx.x % alphabetSize;
    int digit1 = (blockIdx.x - digit0) / alphabetSize;

    int digit2 = blockIdx.y % alphabetSize;
    int digit3 = (blockIdx.y - digit2) / alphabetSize;
    
    int digit4 = blockIdx.z % alphabetSize;
    int digit5 = (blockIdx.z - digit4) / alphabetSize;

    // If word is shorter than six digits, do not execute comparison multiple times for given word
    if (wordLength > 0) currentWord[0] = digit0;
    else if (digit0 != 0) return;

    if (wordLength > 1) currentWord[1] = digit1;
    else if (digit1 != 0) return;

    if (wordLength > 2) currentWord[2] = digit2;
    else if (digit2 != 0) return;

    if (wordLength > 3) currentWord[3] = digit3;
    else if (digit3 != 0) return;

    if (wordLength > 4) currentWord[4] = digit4;
    else if (digit4 != 0) return;

    if (wordLength > 5) currentWord[5] = digit5;
    else if (digit5 != 0) return;

    for (int i = 6; i < wordLength; i++) {
        currentWord[i] = 0;
    }

    // Get all permutations for the remainder of the word
    // Calculate hashes for all permutations
    do {
        indexesToChars(currentWord, currentChars, wordLength, alphabet);
        md5(currentChars, wordLength, digest);

        if (compareTargetHash(targetHash, digest)) {
            printf("Target word: %s\n", currentChars);
            my_strcpy(output, currentChars, wordLength);
        }

    } while (nextWord(currentWord, wordLength));

    free(currentWord);
    free(currentChars);
}

void crackMd5(const char * targetHash, int wordLength) {
    char * h_Output = (char *)malloc(sizeof(char) * wordLength + 1);
    
    // Device memory to store the output word
    char* d_Output;
    hipMalloc(&d_Output, sizeof(char) * wordLength + 1);

    char* d_TargetHash;
    hipMalloc(&d_TargetHash, sizeof(char) * 32);
    hipMemcpy(d_TargetHash, targetHash, sizeof(char) * 32, hipMemcpyHostToDevice);

    char* d_alphabet;
    hipMalloc(&d_alphabet, sizeof(alphabet));
    hipMemcpy(d_alphabet, alphabet, sizeof(alphabet), hipMemcpyHostToDevice);

    const int size = alphabetSize;
    
    dim3 blocks(size * size, size * size, size * size);
    kernelCrackMd5<<<blocks, 1>>>(d_TargetHash, wordLength, d_Output, d_alphabet);

    // Copy the output word from device to host
    hipMemcpy(h_Output, d_Output, sizeof(char) * wordLength + 1, hipMemcpyDeviceToHost);

    hipFree(d_Output);
    hipFree(d_TargetHash);
    hipFree(d_alphabet);
}
